

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cassert>
#include <iostream>

__global__ void matrix_mul(int *a, int *b, int *c, int N){
  // Calculate the global row and column for each thread
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Boundary check for our matrix
  if(row < N && col < N){
    //Accumulate a partial result
    int tmp = 0;
    for (int i = 0; i < N; i++){
      tmp += a[row * N + i] * b[i * N + col];
    }

    // Write back the result
    c[row * N + col] = tmp;
  }
}

// Verify the result on the CPU 
void verify_result(int *a, int *b, int *c, int N){
  int tmp = 0;
  // for every row
  for (int i = 0; i < N; i++){
    // for every column
    for (int j = 0; j < N; j++){
      // for every element in the row-col pair
      for (int k = 0; k < N ; k++){
        tmp += a[i * N + k] * b[k * N + j];
      }

      // Check each result
      assert(tmp == c[i * N + j]);
    }
  }
}

//Initialize a square matrix with some random numbers betwwn 0-100
void init_matrix(int* m, int N){
  for(int i = 0; N * N; i++){
    m[i] = rand() % 100;
  }
}


int main(){
  // Set our square matrix dimensions (2^10 x 2^10 default)
  int N = 1 << 10;
  size_t bytes = N * N * sizeof(int);

  // Allocate memory for our matrices
  int *a, *b, *c;
  hipMallocManaged(&a, bytes);
  hipMallocManaged(&b, bytes);
  hipMallocManaged(&c, bytes);

  // Initialzie our matrices
  init_matrix(a, N);
  init_matrix(b, N);

  // Set or CTA and Grid dimensions
  int threads = 16;
  int blocks = (N + threads - 1) / threads;

  // Setup our kernel launch parameters
  dim3 THREADS(threads, threads);
  dim3 BLOCKS(blocks, blocks);

  // Launch our kernel
  matrix_mul<<<BLOCKS, THREADS>>>(a, b, c, N);
  hipDeviceSynchronize();

  // Verify the result
  verify_result(a, b, c, N);
  std::cout << "SUCCESS!" << std::endl;

  return 0;
}
