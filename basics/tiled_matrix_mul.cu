#include "hip/hip_runtime.h"

#include <cstdlib>
#include <cassert>

__global__ void tiled_sq_mat_mul(float* A, float* B, float* C, int N)
{
    // Details regarding this thread 
    int by = blockIdx.y;
    int bx = blockIdx.x;

    int ty = threadIdx.y;
    int tx = threadIdx.x;

    // Working on C[i, j]
    int i = blockDim.y*by + ty;
    int j = blockDim.x*bx + tx;

    // Allocating shared memory
    __shared__ float sh_a[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sh_a[TILE_WIDTH][TILE_WIDTH];

    // Parallel mat mul
    float value = 0;
    
    // Splitting data into smaller tiles
    for (int phase = 0; phase < N/TILE_WIDTH; phase++){

        // Load tiles into shared memory 
        if ((i < N) && ((phase*TILE_WIDTH*tx)) < N){
            sh_A[ty][tx] = A[(i)*N + phase*TILE_WIDTH+tx]
        } else {
            sh_A[ty][tx] = 0.0f;
        }

        if ((j < N) && ((phase*TILE_WIDTH*ty)) < N){
            sh_B[ty][tx] = B[(phase*TILE_WIDTH + ty)*N+j];
        } else {
            sh_B[ty][tx] = 0.0f;
        }
            __syncthreads();

        // Dot product with data in shared memory
        for (int k = 0; k < TILE_WIDTH; k++){
            value += sh_A[ty][k] * sh_B[k][tx];
        }
        __syncthreads();
    }

    //  Assigning calcualted values

    if ((i<N) && (j<N)){
        C[i*N+j] = value;
    }
}