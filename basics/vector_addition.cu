

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cassert>
#include <iostream>

using namespace std;

// Computes the sum of two arrays 
__global__ void vectorAdd(int *a, int *b, int *c, int N){
  // Calculate the global thread id
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  
  // Range check
  if(tid < N)
    c[tid] = a[tid] + b[tid];
} 

//Verify the vector add computation on the CPU
void verify_solution(int *a, int *b, int *c, int N){
  for (int i = 0; i < N; i++){
    assert(a[i] + b[i] == c[i]);
  }
}

// Initializes array of size "N" with random numbers between 0 and 100
void init_array(int *a, int N){
  for (int i = 0; i < N; i++){
    a[i] = rand() % 100;
  }
}
 
int main (){
  // Set our problem size (Default = 2^20)
  int N = 1 << 20;
  size_t bytes = N * sizeof(bytes);

  // Allocate some memory for inputs/outputs
  int *a, *b, *c; 
  hipMallocManaged(&a, bytes);
  hipMallocManaged(&b, bytes);
  hipMallocManaged(&c, bytes);

  //Initialize our data 
  init_array(a, N);
  init_array(b, N);

  // Initialize our CTA and Grid dmensions
  int THREADS = 256;
  int BLOCKS = (N + THREADS - 1) / THREADS;

  // Call the kernel
  vectorAdd<<<BLOCKS, THREADS>>>(a, b, c, N);
  hipDeviceSynchronize();

  verify_solution(a, b, c, N);
  cout << "SUCCESS!" << endl;

  return 0;
}
